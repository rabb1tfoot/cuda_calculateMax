#include "hip/hip_runtime.h"
﻿

#include <hip/hip_runtime.h>
#ifndef __HIPCC__  
#define __HIPCC__
#endif
#include <hip/device_functions.h>
#include <>


#include <iostream>

#include "cuMax.h"
#include <ctime>

hipError_t MaxAlgo(float *in, int sizeX, int sizeY);
void ToDouble(unsigned char* input, double* output, int size);
void ToFloat(unsigned char* input, float* output, int size);

__device__ void atomicMax(float* address, float value)
{
	if (*address >= value)
	{
		return;
	}

	int* const addressAsI = (int*)address;
	int old = *addressAsI, assumed;

	do
	{
		assumed = old;
		if (__int_as_float(assumed) >= value)
		{
			break;
		}

		old = atomicCAS(addressAsI, assumed, __float_as_int(value));
	} while (assumed != old);
}

__global__ void MaxKernel(float* __restrict__ input, const int* size, float* maxOut, int* maxIdxOut)
{
	float localMax = 0.f;
	int localMaxIdx = 0;

	for (int i = threadIdx.x; i < (*size); i += blockDim.x)
	{
		float val = input[i];

		if (localMax < abs(val))
		{
			localMax = abs(val);
			localMaxIdx = i;
		}
	}

	atomicMax(maxOut, localMax);

	__syncthreads();

	if (*maxOut == localMax)
	{
		*maxIdxOut = localMaxIdx;
	}
}

int main()
{
	std::string filePath = "D:\\Projects\\cuMax\\x64\\Debug\\test1.bmp";

	CImgLoader* manager = new CImgLoader(filePath);
	float* buffer = new float[manager->m_bSize_X * manager->m_bSize_Y];

	ToFloat(manager->m_buffer, buffer, manager->m_bSize_X * manager->m_bSize_Y);



	hipError_t cudaStatus = MaxAlgo(buffer, manager->m_bSize_X, manager->m_bSize_Y);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();

	delete(manager);
	delete(buffer);
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t MaxAlgo(float *in, int size_X, int size_Y)
{
	float *dev_in;
	float *dev_out;
	int *dev_index;
	int *dev_size;

	int* outIndex = new int();
	*outIndex = 0;

	int *size = new int();
	*size = size_X * size_Y;

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);

	// Allocate GPU inputs  .
	cudaStatus = hipMalloc((void**)&dev_in, (*size) * sizeof(float));
	cudaStatus = hipMalloc((void**)&dev_out, (*size) * sizeof(float));
	cudaStatus = hipMalloc((void**)&dev_size, sizeof(int));
	cudaStatus = hipMalloc((void**)&dev_index, sizeof(int));

	// Copy inputs from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_in, in, (*size) * sizeof(float), hipMemcpyHostToDevice);
	cudaStatus = hipMemcpy(dev_size, size, sizeof(int), hipMemcpyHostToDevice);

	LARGE_INTEGER        tFreq, tStart, tEnd;
	float                           tElapsedTime;
	QueryPerformanceFrequency(&tFreq);        // 주파수 측정
	QueryPerformanceCounter(&tStart);
	// Launch a kernel on the GPU with one thread for each element.
	MaxKernel<<<1, 1024 >>>(dev_in, dev_size, dev_out, dev_index);
	QueryPerformanceCounter(&tEnd);
	tElapsedTime = ((tEnd.QuadPart - tStart.QuadPart) / (float)tFreq.QuadPart) * 1000; //ms단위

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(outIndex, dev_index, sizeof(int), hipMemcpyDeviceToHost);

	std::cout << "gpu 연산결과 인덱스 : " << *outIndex << "\n";
	std::cout << "gpu 수행시간 : " << tElapsedTime << " ms\n";

	hipFree(dev_in);
	hipFree(dev_size);
	hipFree(dev_out);
	hipFree(dev_index);

	delete(outIndex);
	delete(size);

	return cudaStatus;
}

void ToDouble(unsigned char* input, double* output, int size)
{
	output = new double[size];
	int maxIdx = 0;
	double maxvalue = 0;

	for (int i = 0; i < size; ++i)
	{
		output[i] = static_cast<double>(input[i]);
		double comp = output[i];
		if (comp > maxvalue)
		{
			maxvalue = comp;
			maxIdx = i;
		}
	}

	int aa = 0;
}

void ToFloat(unsigned char * input, float * output, int size)
{
	int maxIdx = 0;
	float maxvalue = 0;

	for (int i = 0; i < size; ++i)
	{
		output[i] = static_cast<float>(input[i]);
		float comp = output[i];
		if (comp > maxvalue)
		{
			maxvalue = comp;
			maxIdx = i;
		}
	}

	int ab = 0;

	std::cout << "cpu 연산결과 인덱스 : " << maxIdx << "\n";
}
